#include "hip/hip_runtime.h"
#include "unpack.h"

constexpr int features1_size = sizeof(features1_t) / sizeof(DType) / SquareNum;
constexpr int features2_size = sizeof(features2_t) / sizeof(DType) / SquareNum;

__global__ void unpack_features1_kernel(char* p1, short* x1) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int p1_offset = sizeof(packed_features1_t) * 8 * blockIdx.x + threadIdx.x * 81;
	int x1_offset = tid * 81;
#pragma unroll
	for (int i = 0; i < 81; ++i) {
		int j = p1_offset + i;
		// p1[j / 8] >> (j % 8)�ŉ���1bit�ɐݒ肷��l�������Ă���
		// ����1bit�̃}�X�N���s���A�����𕉂ɂ��邱�Ƃ�1�̏ꍇ1byte�̑Sbit��1�ɂ���
		// 0x3c00�Ƙ_���ς���邱�Ƃ�float16��1.0�ɂ���
		x1[x1_offset + i] = (-(short)((p1[j >> 3] >> (j & 7)) & 1)) & 0x3c00;
	}
}

__global__ void unpack_features2_kernel(char* p2, short* x2) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int j = sizeof(packed_features2_t) * 8 * blockIdx.x + threadIdx.x;
	short v = (-(short)((p2[j >> 3] >> (j & 7)) & 1)) & 0x3c00;

	int x2_offset = tid * 81;
#pragma unroll
	for (int i = 0; i < 81; ++i) {
		x2[x2_offset + i] = v;
	}
}

void unpack_features1(const int batch_size, packed_features1_t* p1, features1_t* x1, hipStream_t stream)
{
	unpack_features1_kernel<<<batch_size, features1_size, 0, stream>>>((char*)p1, (short*)x1);
}

void unpack_features2(const int batch_size, packed_features2_t* p2, features2_t* x2, hipStream_t stream)
{
	unpack_features2_kernel<<<batch_size, features2_size, 0, stream>>> ((char*)p2, (short*)x2);
}
